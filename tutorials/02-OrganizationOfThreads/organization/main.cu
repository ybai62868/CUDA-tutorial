#include "hip/hip_runtime.h"


#include <cstdio>


__global__ void print_threadIds() {
    printf("blockIdx.x : %d, blockIdx.y : %d, blockIdx.z : %d , blockDim.x : %d, blockDim.y : %d, gridDim.x : %d, gridDim.y : %d \n",
            blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y,
            gridDim.x, gridDim.y);

    
}


int main(void)
{
    int nx, ny;
    nx = 16;
    ny = 16;
    dim3 block(8, 8);
    dim3 grid(nx / block.x, ny / block.y);

    print_threadIds <<<grid, block>>> ();
    hipDeviceSynchronize();

    hipDeviceReset();


    return 0;
}

