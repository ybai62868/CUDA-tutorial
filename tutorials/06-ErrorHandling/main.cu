#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "cuda_common.cuh"
#include <cstdio>


#include <cstdlib>
#include <ctime>
#include <cstring>


__global__ void sum_array_gpu(int* a, int* b, int* c, int size)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < size) {
      c[gid] = a[gid] + b[gid];
    }
}

void sum_array_cpu(int* a, int* b, int* c, int size)
{
    for ( int i = 0;i < size;i++ ) {
        c[i] = a[i] + b[i];      
    }
}


void compare_arrays(int* a, int* b, int size)
{
    for ( int i = 0;i < size;i++ ) {
      if (a[i] != b[i]) { 
          printf("Arrays are different!");
          return;
      }
    }
    printf("Arrays are same\n");
}


int main(void)
{
    int size = 10000;
    int block_size = 128;

    int NO_BYTES = size * sizeof(int);

    // host pointer
    int* h_a, *h_b, *gpu_results;
    int* h_c;
    h_a = (int*)malloc(NO_BYTES);
    h_b = (int*)malloc(NO_BYTES);
    h_c = (int*)malloc(NO_BYTES);
    gpu_results = (int*)malloc(NO_BYTES);
    hipError_t error;
    
  

    time_t t;
    srand((unsigned)time(&t));
    for (int i = 0;i < size;i++) {
        h_a[i] = (int)(rand() & 0xff);
        h_b[i] = (int)(rand() & 0xff);
    }
    
    sum_array_cpu(h_a, h_b, h_c, size);
    memset(gpu_results, 0, NO_BYTES);


    // device pointer
    int* d_a, *d_b, *d_c;
    error = hipMalloc((int**)&d_a, NO_BYTES);
    if (error != hipSuccess) {
        fprintf(stderr, "Error : %s \n", hipGetErrorString(error));
    }

    hipMalloc((int**)&d_b, NO_BYTES);
    hipMalloc((int**)&d_c, NO_BYTES);

    hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);


    // launching the grid
    dim3 block(block_size);
    dim3 grid((size / block.x) + 1);

    sum_array_gpu<<<grid, block>>> (d_a, d_b, d_c, size);
    hipDeviceSynchronize();


    hipMemcpy(gpu_results, d_c, NO_BYTES, hipMemcpyDeviceToHost);


    // array comparison
    compare_arrays(h_c, gpu_results, size);



    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    

    free(gpu_results);
    hipDeviceReset();


    return 0;

}
