#include "hip/hip_runtime.h"



#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <ctime>

__global__ void mem_trs_test(int* input)
{
    int grid = blockIdx.x * blockDim.x + threadIdx.x;
    printf("tid : %d, gid : %d, value : %d \n", threadIdx.x, grid, input[grid]);

}


int main(void)
{
  int size = 128;
  int byte_size = size * sizeof(int);

  int* h_input = (int*)malloc(byte_size);

  time_t t;
  srand((unsigned)time(&t));
  for ( int i = 0;i < size;i++ ) {
    h_input[i] = (int)(rand() & 0xff);
  }

  int* d_input;
  hipMalloc((void**)&d_input, byte_size);
  hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);
  
  dim3 block(64);
  dim3 grid(2);

  mem_trs_test<<<grid, block>>>(d_input);

  hipDeviceSynchronize();

  hipFree(d_input);
  free(h_input);


  hipDeviceReset();
  return 0;
}
